#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "NeuralNetwork.h"
#include <chrono>




int main()
{
    // 1 - Initiate NeuralNetwork Object
    NeuralNetwork YSA(128, 10);
    // 2 - Load Trainig Samples and Labels
    YSA.ReadTrainingSamples("Train60000_1D_Array.txt");
    YSA.ReadLabels("Labels60000_1D_Array.txt");
    YSA.RandomizeWeights();
    YSA.NormalizeInput();
    // 3 - Start Training
    
    // counterCYCLE
   /* while(!YSA.isTrained())*/

    auto start = std::chrono::high_resolution_clock::now();

    YSA.Train();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double > fp_ms = end - start;
    std::cout << " Gecen Sure :" << fp_ms.count() << "\n";

    //YSA.printOut();
    printf("OK\n");
    // 4 - Loop While Not Trained

    return 0;
}