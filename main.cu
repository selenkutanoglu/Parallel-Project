#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "NeuralNetwork.h"

int main()
{
    // 1 - Initiate NeuralNetwork Object
    NeuralNetwork YSA(128, 10);
    // 2 - Load Trainig Samples and Labels
    YSA.ReadTrainingSamples("Train60000_1D_Array.txt");
    YSA.ReadLabels("Labels60000_1D_Array.txt");
    YSA.RandomizeWeights();
    YSA.NormalizeInput();
    // 3 - Start Training
    YSA.Iterate();
    YSA.printOut();
    printf("OK\n");
    // 4 - Loop While Not Trained

    return 0;
}